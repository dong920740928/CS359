#include "hip/hip_runtime.h"
/*
	*this file exercise matrix multiplication with shared memory and use 
	*the thought of dividing matrix to sub_matrix 
	*/

#include<time.h>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include<hip/hip_runtime_api.h>

#define BLOCK_SIZE 8
#define MATRIX_SIZE 64

typedef struct {
	int width;
	int height;
	float *vals;
} Matrix;

float& GetElement(const Matrix A, int row, int col) {
	return A.vals[row * A.width + col];
}

__device__ float& GetElementKernel(const Matrix A, int row, int col) {
	return A.vals[row * A.width + col];
}

__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C) {
	//__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	//__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
	int range = A.width / BLOCK_SIZE;
	float c_value = 0.0f;
	for (int k = 0; k < range; ++k) {
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[threadIdx.x][threadIdx.y] = GetElementKernel(A, blockIdx.x * BLOCK_SIZE + threadIdx.x, k * BLOCK_SIZE + threadIdx.y);
		Bs[threadIdx.x][threadIdx.y] = GetElementKernel(B, k * BLOCK_SIZE + threadIdx.x, blockIdx.y * BLOCK_SIZE + threadIdx.y);
		
		__syncthreads();

		float tmp = 0.0f;
		for (int block_k = 0; block_k < BLOCK_SIZE; ++block_k) {
			tmp += As[threadIdx.x][block_k] * Bs[block_k][threadIdx.y];
		}
		
		c_value += tmp;
		__syncthreads();
	}

	GetElementKernel(C, blockIdx.x * BLOCK_SIZE + threadIdx.x, blockIdx.y * BLOCK_SIZE + threadIdx.y) = c_value;
}

void MatMulUsual(const Matrix A, const Matrix B, Matrix C) {
	for (int i = 0; i < C.height; ++i) {
		for (int j = 0; j < C.width; ++j) {
			float res = 0.0f;
			for (int k = 0; k < A.width; ++k) {
				res += GetElement(A, i, k) * GetElement(B, k, j);
			}
			GetElement(C, i, j) = res;
		}
	}
}

void checkCUDAError(const char *msg);

int main() {
	size_t memSize = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);
	
	//initialize two matrix
	srand(time(NULL));
	float *valsA = (float*)malloc(memSize);
	float *valsB = (float*)malloc(memSize);
	for (int i = 1; i <= MATRIX_SIZE; ++i) {
		for (int j = 1; j <= MATRIX_SIZE; ++j) {
			valsA[(i - 1) * MATRIX_SIZE + (j - 1)] = (float)(rand()%100);
			valsB[(i - 1) * MATRIX_SIZE + (j - 1)] = (float)(rand()%100);
		}
	}
	Matrix matrixA = {MATRIX_SIZE, MATRIX_SIZE, valsA};
	Matrix matrixB = {MATRIX_SIZE, MATRIX_SIZE, valsB};
	
	//multiplicate with CPU
	float *valsC_CPU = (float*)malloc(memSize);
	Matrix matrixC_CPU = {MATRIX_SIZE, MATRIX_SIZE, valsC_CPU};
	MatMulUsual(matrixA, matrixB, matrixC_CPU);
	
	//multiplicate withGPU
	float *valsC_GPU = (float*)malloc(memSize);
	Matrix matrixC_GPU = {MATRIX_SIZE, MATRIX_SIZE, valsC_GPU};

	//no use
	//	int numBlocks = 8 * 8;
	//int numThreadsPerBlock = MATRIX_SIZE * MATRIX_SIZE / numBlocks;
	
	float *valsA_d, *valsB_d, *valsC_d;
	hipMalloc(&valsA_d, memSize);
	hipMemcpy(valsA_d, valsA, memSize, hipMemcpyHostToDevice);
	hipMalloc(&valsB_d, memSize);
	hipMemcpy(valsB_d, valsB, memSize, hipMemcpyHostToDevice);
	hipMalloc(&valsC_d, memSize);

	Matrix A_d = {MATRIX_SIZE, MATRIX_SIZE, valsA_d};
	Matrix B_d = {MATRIX_SIZE, MATRIX_SIZE, valsB_d};
	Matrix C_d = {MATRIX_SIZE, MATRIX_SIZE, valsC_d};
	
	//launch kernel
	dim3 dimGrid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	MatMulKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);
	
	//block until the device has completed
	hipDeviceSynchronize();
	
	//check errors
	checkCUDAError("kernel invocation");
	
	//data fetch
	hipMemcpy(valsC_GPU, valsC_d, memSize, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy");
	
	//verify the data
	for (int i = 0; i < MATRIX_SIZE; ++i) {
		for (int j = 0; j < MATRIX_SIZE; ++j) {
			assert(GetElement(matrixC_CPU, i, j) == GetElement(matrixC_GPU, i, j));
		}
	}

	hipFree(valsA_d);
	hipFree(valsB_d);
	hipFree(valsC_d);

	free(valsA);
	free(valsB);
	free(valsC_CPU);
	free(valsC_GPU);

	printf("Correct!\n");

	hipProfilerStop();	
	return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

